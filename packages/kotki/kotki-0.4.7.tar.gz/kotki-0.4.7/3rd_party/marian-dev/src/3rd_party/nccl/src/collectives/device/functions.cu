#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "core.h"
#include "collectives.h"
#include "common.h"

#define NCCL_FUNC4(coll, op, dtype) \
  NCCL_COLL_NAME(coll, op, dtype), \
  NCCL_COLL_NAME(coll##LL, op, dtype)  \

// Must be consistent with ncclDataType_t
#define NCCL_FUNCS3A(coll, op) \
  NCCL_FUNC4(coll, op,  i8), \
  NCCL_FUNC4(coll, op,  u8), \
  NCCL_FUNC4(coll, op, i32), \
  NCCL_FUNC4(coll, op, u32), \
  NCCL_FUNC4(coll, op, i64), \
  NCCL_FUNC4(coll, op, u64), \
  NCCL_FUNC4(coll, op, f16), \
  NCCL_FUNC4(coll, op, f32), \
  NCCL_FUNC4(coll, op, f64)
#define NCCL_FUNCS3B(coll, op) \
  NCCL_FUNC4(coll, op,  i8), \
  NCCL_FUNC4(coll, op,  i8), \
  NCCL_FUNC4(coll, op,  i8), \
  NCCL_FUNC4(coll, op,  i8), \
  NCCL_FUNC4(coll, op,  i8), \
  NCCL_FUNC4(coll, op,  i8), \
  NCCL_FUNC4(coll, op,  i8), \
  NCCL_FUNC4(coll, op,  i8), \
  NCCL_FUNC4(coll, op,  i8)

// Must be consistent with ncclRedOp_t
#define NCCL_FUNCS2A(coll) \
  NCCL_FUNCS3A(coll, sum ), \
  NCCL_FUNCS3A(coll, prod), \
  NCCL_FUNCS3A(coll, max ), \
  NCCL_FUNCS3A(coll, min )
#define NCCL_FUNCS2B(coll) \
  NCCL_FUNCS3B(coll, copy), \
  NCCL_FUNCS3B(coll, copy), \
  NCCL_FUNCS3B(coll, copy), \
  NCCL_FUNCS3B(coll, copy)

// Must be consistent with ncclColl_t
#define NCCL_FUNCS() { \
  NCCL_FUNCS2B(ncclBroadcast), \
  NCCL_FUNCS2A(ncclReduce), \
  NCCL_FUNCS2B(ncclAllGather), \
  NCCL_FUNCS2A(ncclReduceScatter), \
  NCCL_FUNCS2A(ncclAllReduce) }

// Must be consistent with the ncclFuncSet enum
__device__ ncclKern_t ncclFuncs[ncclCollCount*ncclNumOps*ncclNumTypes*2] = {
  NCCL_FUNCS2B(ncclBroadcast),
  NCCL_FUNCS2A(ncclReduce),
  NCCL_FUNCS2B(ncclAllGather),
  NCCL_FUNCS2A(ncclReduceScatter),
  NCCL_FUNCS2A(ncclAllReduce)
};
